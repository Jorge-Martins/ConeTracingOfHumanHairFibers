#include "hip/hip_runtime.h"
#pragma once

#ifndef _RAYTRACER_KERNEL_CU_
#define _RAYTRACER_KERNEL_CU_


#include <cfloat>

#include "BVH.cuh"

//size ray array
__device__ int const raysSize = 2 << (MAX_DEPTH - 1);

//reflection and refraction arrays 
__device__ int const sizeRRArrays = (2 << (MAX_DEPTH - 1)) - 1;

//size local array
__device__ int const raysPerPixel = (2 << MAX_DEPTH) - 1;



__device__
bool findShadow(int **d_shapes, uint *d_shapeSizes, Ray feeler) {
    bool intersectionFound = false;
    //tmp debug
    float distance = FLT_MAX;
    for(uint shapeType = 0; shapeType < nShapes; shapeType++) {
        for (uint i = 0; i < d_shapeSizes[shapeType]; i++) {
            if(shapeType == sphereIndex) {
                SphereNode *sphereNode = (SphereNode*) d_shapes[shapeType];

                SphereNode *node = &sphereNode[i];
                intersectionFound = AABBIntersection(feeler, node->min, node->max, &distance);

                if(intersectionFound) {
                    intersectionFound = intersection(feeler, nullptr, node->shape);
                }
               
            } else if(shapeType == cylinderIndex) {
                CylinderNode *cylinderNode = (CylinderNode*) d_shapes[shapeType];
                uint leafOffset = d_shapeSizes[shapeType] - 1;

                CylinderNode *node = &cylinderNode[leafOffset + i];
                if(node->type == AABB) {
                    intersectionFound = AABBIntersection(feeler, node->min, node->max, &distance);
                } else {
                    intersectionFound = OBBIntersection(feeler, node->min, node->max, 
                                                        node->matrix, node->translation, &distance);
                }

                if(intersectionFound) {
                    intersectionFound = intersection(feeler, nullptr, node->shape);
                }

            } else if(shapeType == triangleIndex) {
                TriangleNode *triangleNode = (TriangleNode*) d_shapes[shapeType];

                TriangleNode *node = &triangleNode[i];
                intersectionFound = AABBIntersection(feeler, node->min, node->max, &distance);

                if(intersectionFound) {
                    intersectionFound = intersection(feeler, nullptr, node->shape);
                }

            } else if(shapeType == planeIndex) {
                Plane *plane = (Plane*) d_shapes[shapeType];
                intersectionFound = intersection(feeler, nullptr, plane[i]);

            } else {
                return false;
            }

            if(intersectionFound) {
                return true;
            } 
	    }
    }

    return intersectionFound;
}

__device__
bool nearestIntersect(int **d_shapes, uint *d_shapeSizes, Ray ray, RayIntersection *out) {
	RayIntersection minIntersect(FLT_MAX, make_float3(0.0f), make_float3(0.0f));
	bool minIntersectionFound = false, intersectionFound = false;

	RayIntersection curr = minIntersect;
    //tmp debug
    float distance = FLT_MAX;
    for(uint shapeType = 0; shapeType < nShapes; shapeType++) {
        for (uint i = 0; i < d_shapeSizes[shapeType]; i++) {
            if(shapeType == sphereIndex) {
                SphereNode *sphereNode = (SphereNode*) d_shapes[shapeType];

                SphereNode *node = &sphereNode[i];
                intersectionFound = AABBIntersection(ray, node->min, node->max, &distance);

                if(intersectionFound) {
                    intersectionFound = intersection(ray, &curr, node->shape);
                }
               
            } else if(shapeType == cylinderIndex) {
                CylinderNode *node = nullptr;
                
                CylinderNode *bvh = (CylinderNode*) d_shapes[shapeType];

                bool bvhHit = traverse(node, bvh, d_shapeSizes[shapeType], ray);

                if(bvhHit) {
                    intersectionFound = intersection(ray, &curr, node->shape);
                }
                
                break;
                //uint leafOffset = d_shapeSizes[shapeType] - 1;

                
                /*CylinderNode *node = &cylinderNode[leafOffset + i];
                if(node->type == AABB) {
                    intersectionFound = AABBIntersection(ray, node->min, node->max);
                } else {
                    intersectionFound = OBBIntersection(ray, node->min, node->max, 
                                                        node->matrix, node->translation);
                }

                if(intersectionFound) {
                    intersectionFound = intersection(ray, &curr, node->shape);
                }*/

            } else if(shapeType == triangleIndex) {
                TriangleNode *triangleNode = (TriangleNode*) d_shapes[shapeType];

                TriangleNode *node = &triangleNode[i];
                intersectionFound = AABBIntersection(ray, node->min, node->max, &distance);

                if(intersectionFound) {
                    intersectionFound = intersection(ray, &curr, node->shape);
                }

            } else if(shapeType == planeIndex) {
                Plane *plane = (Plane*) d_shapes[shapeType];
                intersectionFound = intersection(ray, &curr, plane[i]);

            } else {
                return false;
            }

		    if (intersectionFound) {
                if (curr.distance < minIntersect.distance) {
                    minIntersectionFound = true;
				    minIntersect = curr;
			    }
		    }
	    }
    }
    
	if (minIntersectionFound) {
		*out = minIntersect;
	}
	return minIntersectionFound;
}


__device__
int rayIndex(int index) {
    return index % raysSize;
}

__device__
float3 refract(float3 inDir, float3 normal, float eta) {
    float cosi = dot(-inDir, normal);
    float cost2 = 1.0f - eta * eta * (1.0f - cosi*cosi);
    float3 t = eta*inDir + ((eta*cosi - sqrt(abs(cost2))) * normal);

    if(cost2 > 0) {
        return t;
    } 

    return make_float3(0.0f);
}

__device__
float3 computeTransmissionDir(float3 inDir, float3 normal, float beforeIOR, float afterIOR) {
	return refract(inDir, normal, beforeIOR / afterIOR);
}

__device__
float3 rayTracing(int **d_shapes, uint *d_shapeSizes, Light* lights, uint lightSize, float3 backcolor, 
                 float3 rayOrigin, float3 rayDirection, Ray *ray, float3* locals,
                 float3* reflectionCols, float3* refractionCols, uint offset) {

    uint rayOffset = offset * raysSize;                 
    uint localsOffset = offset * raysPerPixel;
    uint rrOffset = offset * sizeRRArrays;
    Ray feeler = Ray();

    ray[rayOffset].update(rayOrigin, rayDirection);
    
    RayIntersection intersect;
    int level;
    float3 blackColor = make_float3(0.0f);
    
    for(int rayN = 0; rayN < raysPerPixel; rayN++) {
        //skip secundary rays that don't exist
        if(!ray[rayOffset + rayIndex(rayN)].exists) {
            if(rayN < sizeRRArrays) {
                reflectionCols[rrOffset + rayN] = blackColor;
                refractionCols[rrOffset + rayN] = blackColor;
                level = 2 * rayN;
                ray[rayOffset + rayIndex(level + 1)].exists = false;
                ray[rayOffset + rayIndex(level + 2)].exists = false;
            }
            continue;
        }

	    bool foundIntersect = nearestIntersect(d_shapes, d_shapeSizes, ray[rayOffset + rayIndex(rayN)], &intersect);

	    if (!foundIntersect) {
            if(rayN == 0) {
                return backcolor;
            }

            locals[localsOffset + rayN] = backcolor;

            if(rayN < sizeRRArrays) {
                reflectionCols[rrOffset + rayN] = blackColor;
                refractionCols[rrOffset + rayN] = blackColor;
                level = 2 * rayN;
                ray[rayOffset + rayIndex(level + 1)].exists = false;
                ray[rayOffset + rayIndex(level + 2)].exists = false;
            }
            continue;
        } 

        Material mat = intersect.shapeMaterial;
         
        // local illumination
	    locals[localsOffset + rayN] = blackColor;
	    for(uint li = 0; li < lightSize; li++) {
		    float3 feelerDir = normalize(lights[li].position - intersect.point);
            feeler.update(intersect.point, feelerDir);
            
            bool inShadow = findShadow(d_shapes, d_shapeSizes, feeler);
                
		    if(!inShadow) {
                float3 reflectDir = reflect(-feelerDir, intersect.normal);
                float Lspec = powf(fmaxf(dot(reflectDir, -ray[rayOffset + rayIndex(rayN)].direction), 0.0f), 
                                         mat.shininess);
                float Ldiff = fmaxf(dot(feelerDir, intersect.normal), 0.0f);

			    
                locals[localsOffset + rayN] +=  (Ldiff * mat.color * mat.Kdiffuse + mat.color * Lspec * mat.Kspecular) * lights[li].color;
		    }
	    }
    
        if(rayN < sizeRRArrays) {
            reflectionCols[rrOffset + rayN] = blackColor;
            refractionCols[rrOffset + rayN] = blackColor;
            level = 2 * rayN;
            ray[rayOffset + rayIndex(level + 1)].exists = false;
            ray[rayOffset + rayIndex(level + 2)].exists = false;
            // reflection
            level = 2 * rayN + 1;
	        if(mat.Kspecular > 0.0f) {
                float3 reflectDir = reflect(ray[rayOffset + rayIndex(rayN)].direction, intersect.normal);
		        ray[rayOffset + rayIndex(level)].update(intersect.point, reflectDir);
                reflectionCols[rrOffset + rayN] = mat.color * mat.Kspecular;
            }

	        // transmission
            level = 2 * rayN + 2;
	        if(mat.transparency > 0.0f) {
		        float ior1, ior2;
		        if(intersect.isEntering) {
			        ior1 = 1.0f;
			        ior2 = mat.ior;
		        }
		        else {
			        ior1 = mat.ior;
			        ior2 = 1.0f;
		        }
		        float3 refractionDir = computeTransmissionDir(ray[rayOffset + rayIndex(rayN)].direction, intersect.normal, ior1, ior2);
		            
                if (!equal(length(refractionDir), 0.0f)) {
			        ray[rayOffset + rayIndex(level)].update(intersect.point, refractionDir);
                    refractionCols[rrOffset + rayN] = mat.color * mat.transparency;
                }
	        }
        }
    }

    int startLevel = sizeRRArrays - 1;
    int rrLevel = -2;

    if(MAX_DEPTH > 2) {
        rrLevel += 2 << (MAX_DEPTH - 2);
    }

    for(int i = startLevel; i >= 0 && i > rrLevel; i--) {
        level = 2 * i;       
        reflectionCols[rrOffset + i] *= locals[localsOffset + level + 1];
        refractionCols[rrOffset + i] *= locals[localsOffset + level + 2];
    }

    for(int i = rrLevel; i >= 0; i--) {
        level = 2 * i;
        locals[localsOffset + level + 1] += reflectionCols[rrOffset + level + 1] + refractionCols[rrOffset + level + 1];
        locals[localsOffset + level + 2] += reflectionCols[rrOffset + level + 2] + refractionCols[rrOffset + level + 2];
        
        reflectionCols[rrOffset + i] *= locals[localsOffset + level + 1];
        refractionCols[rrOffset + i] *= locals[localsOffset + level + 2];
    }

    return locals[localsOffset] + reflectionCols[rrOffset] + refractionCols[rrOffset];
}


__global__
void drawScene(int **d_shapes, uint *d_shapeSizes, Light *lights, uint lightSize, float3 backcolor, int resX,
               int resY, int res_xy, float width, float height, float atDistance, float3 xe, float3 ye, 
               float3 ze, float3 from, float3 *d_output, Ray* ray, float3* d_locals, 
               float3* d_reflectionCols, float3* d_refractionCols) {

    uint x = blockIdx.x * blockDim.x + threadIdx.x;
    uint y = blockIdx.y * blockDim.y + threadIdx.y;

    uint index = y * (resX * SUPER_SAMPLING) + x;

    if(index < res_xy * SUPER_SAMPLING * SUPER_SAMPLING) {
        float3 zeFactor = -ze * atDistance; 
        float3 yeFactor = ye * height * ((y + 0.5f) / (float)(resY * SUPER_SAMPLING) - 0.5f);
	    float3 xeFactor = xe * width * ((x + 0.5f) / (float)(resX * SUPER_SAMPLING) - 0.5f);

	    float3 direction = normalize(zeFactor + yeFactor + xeFactor);
	
        float3 color = rayTracing(d_shapes, d_shapeSizes, lights, lightSize, backcolor, from, direction, 
                                  ray, d_locals, d_reflectionCols, d_refractionCols, index);
    
        if(SUPER_SAMPLING > 1) {
            index = (uint)(y / (float)SUPER_SAMPLING) * resX + (uint)(x / (float)SUPER_SAMPLING);
            d_output[index] += color; // (SUPER_SAMPLING * SUPER_SAMPLING);
        } else {
            d_output[index] = color;
        }
    }
}

__global__
void clearImage(float3 *d_output, float3 value, int resX, int res) {
    uint x = blockIdx.x * blockDim.x + threadIdx.x;
    uint y = blockIdx.y * blockDim.y + threadIdx.y;

    uint index = y * resX + x;

    if(index < res) {
        d_output[index] = value;
    }
}

void deviceClearImage(float3 *d_output, float3 value, int resX, int resY, dim3 gridSize, dim3 blockSize) {
    clearImage<<<gridSize, blockSize>>>(d_output, make_float3(0.0f), resX, resX * resY);
}

void deviceDrawScene(int **d_shapes, uint *d_shapeSizes, Light* lights, uint lightSize, float3 backcolor, 
                     int resX, int resY, float width, float height, float atDistance, float3 xe, float3 ye, 
                     float3 ze, float3 from, float3 *d_output, dim3 ssgridSize, dim3 blockSize, Ray* ray,
                     float3* d_locals, float3* d_reflectionCols, float3* d_refractionCols) {
    
    int res_xy = resX * resY;
    drawScene<<<ssgridSize, blockSize>>>(d_shapes, d_shapeSizes, lights, lightSize, backcolor, resX, resY,
                                       res_xy, width, height, atDistance, xe, ye, ze, from, d_output, ray,
                                       d_locals, d_reflectionCols, d_refractionCols);

}


void deviceBuildBVH(CylinderNode *bvh, uint nObjects, dim3 gridSize, dim3 blockSize) {

    buildBVH<<<gridSize, blockSize>>>(bvh, nObjects);

    computeBVHBB<<<gridSize, blockSize>>>(bvh, nObjects);
}
#endif