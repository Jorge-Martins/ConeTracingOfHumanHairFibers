#include "hip/hip_runtime.h"
#pragma once

#ifndef _RAYTRACER_KERNEL_CU_
#define _RAYTRACER_KERNEL_CU_

#include <hip/hip_vector_types.h>
#include <vector_functions.h>
#include <math_functions.h>

#include <cfloat>

#include "Scene.h"


__device__
bool equal(float f1, float f2) {
	float diffAbs = abs(f1 - f2);
	return diffAbs < FLOAT_EPSILON;
}


__device__
bool Sphere::intersection(Ray ray, RayIntersection *out) {
    float d_2, r_2, b, c, t;

	float3 rayOrig = ray.origin();
	float xs_xr = _x - rayOrig.x;
	float ys_yr = _y - rayOrig.y;
	float zs_zr = _z - rayOrig.z;

	r_2 = _r * _r;
	d_2 = (xs_xr * xs_xr) + (ys_yr * ys_yr) + (zs_zr * zs_zr);

	if (equal(d_2, r_2)) {
		return false;
	}
	else {
		float3 rayDir = ray.direction();
		b = rayDir.x * xs_xr + rayDir.y * ys_yr + rayDir.z * zs_zr;

		if (d_2 > r_2 && b < 0.0f) {		
			return false;
		}

		c = b*b - d_2 + r_2;

		if (c < 0.0f) {
			return false;
		}

		if (d_2 > r_2) {
			t = b - sqrtf(c);
		}
		else {
			t = b + sqrtf(c);
		}
		if (out != nullptr) {
			out->_point = make_float3(rayOrig.x + rayDir.x * t,
				                        rayOrig.y + rayDir.y * t,
				                        rayOrig.z + rayDir.z * t);

			out->_normal = make_float3((out->_point.x - _x) / _r,
				                        (out->_point.y - _y) / _r,
				                        (out->_point.z - _z) / _r);

			bool entering = true;
			if (d_2 < r_2) {
				out->_normal = out->_normal * -1.0f;
				entering = false;
			}
			out->_shape = this;
			out->_distance = t;
			out->_isEntering = entering;
		}
		return true;
	}
}

__device__
bool Cylinder::intersection(Ray ray, RayIntersection *out) {
    //TODO
    return false;
}


__device__
bool nearestIntersect(Sphere *shapes, size_t shapeSize, Ray ray, RayIntersection *out) {
	RayIntersection minIntersect((float)FLT_MAX, make_float3(0.0f), make_float3(0.0f));
	bool intersectionFound = false;

	RayIntersection curr = minIntersect;
    for (size_t i = 0; i < shapeSize; i++) {
		if (shapes[i].intersection(ray, &curr)) {
            if (curr._distance < minIntersect._distance) {
				intersectionFound = true;
				minIntersect = curr;
			}
		}
	}

	if (intersectionFound) {
		*out = minIntersect;
	}
	return intersectionFound;
}

__device__
void compensatePrecision(Ray &ray) {
	ray.origin() += ray.direction() * ACNE_EPSILON;
}

__device__
float3 refract(float3 inDir, float3 normal, float eta) {
    float cosi = dot(-inDir, normal);
    float cost2 = 1.0f - eta * eta * (1.0f - cosi*cosi);
    float3 t = eta*inDir + ((eta*cosi - sqrt(abs(cost2))) * normal);

    if(cost2 > 0) {
        return t;
    } 

    return make_float3(0.0f);
}

__device__
float3 computeTransmissionDir(float3 inDir, float3 normal, float beforeIOR, float afterIOR) {
	return refract(inDir, normal, beforeIOR / afterIOR);
}

__device__
Color rayTracing(Sphere* shapes, size_t shapeSize, Light* lights, size_t lightSize, Color backcolor, Ray ray, float ior) {

    Ray refractedRay = Ray();
    Ray reflectedRay = Ray();
    Ray feeler = Ray();

    float3 locals[MAX_DEPTH];
    float3 reflectionCols[MAX_DEPTH];
    float3 refractionCols[MAX_DEPTH];

    RayIntersection intersect;
    Ray activeRay;

    for(int depth = 0; depth < MAX_DEPTH; depth++) {

	    bool foundIntersect = nearestIntersect(shapes, shapeSize, ray, &intersect);

	    if (!foundIntersect) {
		    return backcolor;
        }

        Material mat = intersect._shape->material();
    
        // local illumination
	    locals[depth] = make_float3(0.0f);
	    for(size_t li = 0; li < lightSize; li++) {
		    float3 feelerDir = normalize(lights[li].position() - intersect._point);
		    feeler.update(intersect._point, feelerDir);
		    compensatePrecision(feeler);

		    bool inShadow = false;
		    for(size_t si = 0; si < shapeSize; si++) {
			    if (shapes[si].intersection(feeler, nullptr)) {
				    inShadow = true;
				    break;
			    }
		    }
		    if(!inShadow) {
			    float diff = fmax(dot(feelerDir, intersect._normal), 0.0f);
			    float3 reflectDir = reflect(-feelerDir, intersect._normal);
			    float spec = powf(fmax(dot(reflectDir, -ray.direction()), 0.0f), mat.shininess());

			    float3 seenColor = mat.color().color() * lights[li].color().color();
			    locals[depth] += seenColor * (diff * mat.diffuse() + spec * mat.specular());
		    }
	    }
    

        if(depth < MAX_DEPTH) {
	        // reflection
	        if(mat.specular() > 0.0f) {
		        reflectedRay.update(intersect._point, reflect(ray.direction(), intersect._normal));
		        compensatePrecision(reflectedRay);

                reflectionCols[depth] = mat.color().color() * mat.specular();
                //rayTracing(shapes, shapeSize, lights, lightSize, backcolor, reflectedRay, depth - 1, mat.refraction());
	        }

	        // transmission
	        if(mat.transparency() > 0.0f) {
		        float ior1, ior2;
		        if(intersect._isEntering) {
			        ior1 = 1.0f;
			        ior2 = mat.refraction();
		        }
		        else {
			        ior1 = mat.refraction();
			        ior2 = 1.0f;
		        }
		        float3 refractionDir = computeTransmissionDir(ray.direction(), intersect._normal, ior1, ior2);
		        if (!equal(length(refractionDir), 0.0f)) {
			        refractedRay.update(intersect._point, refractionDir);
			        compensatePrecision(refractedRay);
			        
                    refractionCols[depth] = mat.color().color() * mat.transparency();
                    //rayTracing(shapes, shapeSize, lights, lightSize, backcolor, refractedRay, depth - 1, mat.refraction());
		        }
	        }

        } else {
            reflectionCols[depth] = refractionCols[depth] = make_float3(0.0f);
        }
    }

    int i = MAX_DEPTH - 1;
    //locals[i] += reflectionCols[i] + refractionCols[i];

    for(int i = MAX_DEPTH - 2; i >= 0; i--) {
        locals[i] += reflectionCols[i] + refractionCols[i];
        locals[i] += locals[i+1];
    }

	return Color(locals[0]);
}


__global__
void drawScene(Sphere *shapes, size_t shapeSize, Light* lights, size_t lightSize, Color backcolor, int resX,
               int resY, float width, float height, float atDistance, float3 xe, float3 ye, 
               float3 ze, float3 from, float3 *d_output) {

    uint x = blockIdx.x * blockDim.x + threadIdx.x;
    uint y = blockIdx.y * blockDim.y + threadIdx.y;

    float zeFactor = atDistance;
	float yeFactor = height * ((y + 0.5f) / resY - 0.5f);
	float xeFactor = width * ((x + 0.5f) / resX - 0.5f);

	float3 direction = normalize(zeFactor * ze + yeFactor * ye + xeFactor * xe);
	Ray ray = Ray(from, direction);

    Color color = rayTracing(shapes, shapeSize, lights, lightSize, backcolor, ray, 1.0);

    d_output[y * resX + x] = normalize(color.color());

}

void deviceDrawScene(Sphere *shapes, size_t shapeSize, Light* lights, size_t lightSize, Color backcolor, 
                     int resX, int resY, float width, float height, float atDistance, float3 xe, float3 ye, 
                     float3 ze, float3 from, float3 *d_output, dim3 gridSize, dim3 blockSize) {

    drawScene<<<gridSize, blockSize>>>(shapes, shapeSize, lights, lightSize, backcolor, resX, resY,
                                       width, height, atDistance, xe, ye, ze, from, d_output);


}


#endif