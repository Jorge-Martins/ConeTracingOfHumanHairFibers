#include "hip/hip_runtime.h"
#pragma once

#ifndef _RAYTRACER_KERNEL_CU_
#define _RAYTRACER_KERNEL_CU_

#include "AOITHair.cuh"
#include <thrust/random.h>

#define rtStackSize (2 * MAX_DEPTH)
#define STOP_IMPORTANCE 0.04

__device__
float2 cudaRandom(thrust::default_random_engine &rng) {
    return make_float2((float)rng() ,
                       (float)rng());
}

__device__
float haltonSequance(int index, int base) {
       float result = 0;
       float f = 1;
      
       for(int i = index; i > 0; i = floorf(i / base)) {
           f = f / base;
           result = result + f * (i % base);
           
       }

       return result;
}

__device__
float3 printRayHairIntersections(int rayHairIntersections, float3 finalColor, int nRays) {
    if(rayHairIntersections == 0) {
        return finalColor;
    }

    if(rayHairIntersections < 5 * nRays) {
        return make_float3(0, 0, 10);

    } else if (rayHairIntersections < 10 * nRays) {
        return make_float3(0, 10, 0);

    } else if(rayHairIntersections < 20 * nRays) {
        return make_float3(10, 10, 0);

    } else if(rayHairIntersections < 40 * nRays) {
        return make_float3(10, 0, 0);

    }

    return make_float3(0.5, 0, 0);
}


__device__
float3 refract(float3 inDir, float3 normal, float eta) {
    float cosi = dot(-inDir, normal);
    float k = 1.0f - eta * eta * (1.0f - cosi*cosi);

    if(k > 0) {
        return eta*inDir + (eta*cosi - sqrtf(k)) * normal;
    } 

    return make_float3(0.0f);
}

__device__
float3 computeTransmissionDir(float3 inDir, float3 normal, float beforeIOR, float afterIOR) {
	return refract(inDir, normal, beforeIOR / afterIOR);
}





__device__
float3 rayTracing(int **d_shapes, uint *d_shapeSizes, Light* lights, uint lightSize, float3 backcolor, 
                 float3 rayOrigin, float3 rayDirection, RayInfo *globalRayInfo, float3 *globalColors, 
                 unsigned char *globalColorContributionType, uint offset, IntersectionLstItem *globalIntersectionLst,
                 RayIntersection *hairIntersectionLst) {

    RayInfo *rayInfoStack = &globalRayInfo[offset * rtStackSize];

    float3 *localsStack = &globalColors[3 * offset * rtStackSize];
    float3 *reflectionColsStack = &localsStack[rtStackSize];
    float3 *refractionColsStack = &reflectionColsStack[rtStackSize];

    #ifdef AT_SHADOWS
    IntersectionLstItem *atShadowintersectionLst = &globalIntersectionLst[INTERSECTION_LST_SIZE * offset];

    #else 
    IntersectionLstItem *atShadowintersectionLst = nullptr;
    #endif

    unsigned char *colorContributionType = &globalColorContributionType[offset * rtStackSize];

    Ray ray = Ray();

    int rayIndex = 0;
    int colorsIndex = 0;
    int contributionIndex = 0;
    RayInfo info;
    RayIntersection intersect;

    rayInfoStack[rayIndex++].update(rayOrigin, rayDirection);
    float3 blackColor = make_float3(0.0f);
    float3 colorAux;

    int rayHairIntersections = 0;

    int nRays = 0;
    //bool foundIntersect;
    //bool computeColor = false;
    Material mat;

    while(rayIndex > 0) {
        reflectionColsStack[colorsIndex] = blackColor;
        refractionColsStack[colorsIndex] = blackColor;
        nRays++;
        info = rayInfoStack[--rayIndex];
        colorContributionType[contributionIndex++] = info.type;
        ray.update(info.origin, info.direction);
        
        #ifdef AT_HAIR
        localsStack[0] = computeHairAT(d_shapes, d_shapeSizes, lights, lightSize, ray, atShadowintersectionLst,
                                       hairIntersectionLst, backcolor, 100.0f, rayHairIntersections);
        
        #else

        foundIntersect = false;
        #ifdef GENERAL_INTERSECTION 
        foundIntersect = nearestIntersect(d_shapes, d_shapeSizes, ray, &intersect, rayHairIntersections);
        #else
	    foundIntersect = cylNearestIntersect(d_shapes, d_shapeSizes, ray, &intersect, rayHairIntersections);
        #endif

	    if (!foundIntersect) {     
            localsStack[colorsIndex] = backcolor;
            computeColor = true;
            
        } else {      
            mat = intersect.shapeMaterial;

            // local illumination
            colorAux = blackColor;
	        for(uint li = 0; li < lightSize; li++) {
                #ifndef SOFT_SHADOWS
                colorAux += computeShadows(d_shapes, d_shapeSizes, lights, ray, intersect,
                                           li, normalize(lights[li].position - intersect.point),
                                           intersectionLst);
                    
                #else
                colorAux += computeSoftShadows(d_shapes, d_shapeSizes, lights, ray, intersect,
                                               li, normalize(lights[li].position - intersect.point),
                                               intersectionLst);
                #endif
	        }
            
            localsStack[colorsIndex] = colorAux;
     
            computeColor = true;
            if(info.depth < MAX_DEPTH) {
                // reflection
                colorAux = blackColor;
	            if(mat.Kspecular > EPSILON && info.importance > STOP_IMPORTANCE) {
                    float3 reflectDir = reflect(ray.direction, intersect.normal);
                    float importance = info.importance * fminf(length(mat.color) * 0.66f, 1.0f) * mat.Kspecular;
                    rayInfoStack[rayIndex++].update(intersect.point, reflectDir, REFLECTED, 
                                                    info.depth + 1, importance);
                    colorAux = mat.color * mat.Kspecular;
                    computeColor = false;
                }
                reflectionColsStack[colorsIndex] = colorAux;

	            // transmission
                colorAux = blackColor;
                if(mat.transparency > EPSILON && info.importance > STOP_IMPORTANCE) {
		            float ior1, ior2, importance;
		            if(intersect.isEntering) {
			            ior1 = 1.0f;
			            ior2 = mat.ior;
                        importance = mat.transparency;

		            } else {
			            ior1 = mat.ior;
			            ior2 = 1.0f;
                        importance = 1.0f;
		            }
		            float3 refractionDir = computeTransmissionDir(ray.direction, intersect.normal, ior1, ior2);
		            
                    if (length(refractionDir) > EPSILON) {
                        colorAux = mat.color * importance;
                        importance *= info.importance * fminf(length(mat.color) * 0.66f, 1.0f);
			            rayInfoStack[rayIndex++].update(intersect.point, refractionDir, REFRACTED, 
                                                        info.depth + 1, importance);
                        computeColor = false;
                    }
	            }
                refractionColsStack[colorsIndex] = colorAux;

                if(!computeColor) {
                    colorsIndex++;
                }
            }
        }

        #ifndef PRINT_N_INTERSECTIONS
        if(computeColor && info.type != PRIMARY) {
            if(info.type == REFLECTED) {
                unsigned char type;
                int prevColorsIndex;
                while(1) {
                    colorAux = localsStack[colorsIndex] + reflectionColsStack[colorsIndex] + refractionColsStack[colorsIndex];

                    type = colorContributionType[--contributionIndex];
                    prevColorsIndex = colorsIndex - 1;
                    if(type == REFLECTED) {
                        reflectionColsStack[prevColorsIndex] *= colorAux;
                        
                    } else if(type == REFRACTED) {
                        refractionColsStack[prevColorsIndex] *= colorAux;
                        
                    } 

                    if(prevColorsIndex <= rayIndex) {
                        break;
                    } 

                    colorsIndex--;
                }

            } else {
                colorAux = localsStack[colorsIndex] + reflectionColsStack[colorsIndex] + refractionColsStack[colorsIndex];
                refractionColsStack[colorsIndex - 1] *= colorAux;
                contributionIndex--;
            }
        }
        #endif
        computeColor = false;

        #endif
    }

    #ifndef PRINT_N_INTERSECTIONS
    colorAux = localsStack[0] + reflectionColsStack[0] + refractionColsStack[0];
    
    #else
    colorAux = make_float3(rayHairIntersections, nRays, 0.0f);
    
    #endif
    
    return colorAux;
}

__device__
float3 naiveSupersampling(int **d_shapes, uint *d_shapeSizes, Light *lights, uint lightSize, float3 backcolor, 
                          float3 xe, float3 ye, float3 zeFactor, float3 from, RayInfo* rayInfo, float3* d_colors, 
                          unsigned char *d_colorContributionType, uint index, uint x, uint y, int resX, int resY,
                          IntersectionLstItem *d_intersectionLst, RayIntersection *hairIntersectionLst) {

    float3 direction, color = make_float3(0.0f), yeFactor, xeFactor;
    for(int sx = 0; sx < SUPER_SAMPLING; sx++) {
        for(int sy = 0; sy < SUPER_SAMPLING; sy++) {
            yeFactor = ye * ((y + (sy + 0.5f) * SUPER_SAMPLING_F) / (float)resY - 0.5f);
            xeFactor = xe * ((x + (sx + 0.5f) * SUPER_SAMPLING_F) / (float)resX - 0.5f);

            direction = normalize(zeFactor + yeFactor + xeFactor);

            color += SUPER_SAMPLING_2_F * rayTracing(d_shapes, d_shapeSizes, lights, lightSize, backcolor, from, direction, 
                                                     rayInfo, d_colors, d_colorContributionType, index, d_intersectionLst,
                                                     hairIntersectionLst);
        }
    }

    return color;
}

__device__
float3 naiveRdmSupersampling(int **d_shapes, uint *d_shapeSizes, Light *lights, uint lightSize, float3 backcolor, 
                             float3 xe, float3 ye, float3 zeFactor, float3 from, RayInfo* rayInfo, float3* d_colors, 
                             unsigned char *d_colorContributionType, uint index, uint x, uint y, int resX, int resY,
                             long seed, IntersectionLstItem *d_intersectionLst, RayIntersection *hairIntersectionLst) {

    thrust::default_random_engine rng(seed + index);
    thrust::uniform_real_distribution<float> uniDist;
    rng.discard(2 * index);

    float3 direction, color = make_float3(0.0f), yeFactor, xeFactor;
    for(int sx = 0; sx < SUPER_SAMPLING; sx++) {
        for(int sy = 0; sy < SUPER_SAMPLING; sy++) {
            yeFactor = ye * ((y + (sy + uniDist(rng)) * SUPER_SAMPLING_F) / (float)resY - 0.5f);
            xeFactor = xe * ((x + (sx + uniDist(rng)) * SUPER_SAMPLING_F) / (float)resX - 0.5f);

            direction = normalize(zeFactor + yeFactor + xeFactor);

            color += SUPER_SAMPLING_2_F * rayTracing(d_shapes, d_shapeSizes, lights, lightSize, backcolor, from, direction, 
                                                     rayInfo, d_colors, d_colorContributionType, index, d_intersectionLst,
                                                     hairIntersectionLst);
        }
    }

    return color;
}

__device__
float3 stocasticSupersampling(int **d_shapes, uint *d_shapeSizes, Light *lights, uint lightSize, float3 backcolor, 
                              float3 xe, float3 ye, float3 zeFactor, float3 from, RayInfo* rayInfo, float3* d_colors, 
                              unsigned char *d_colorContributionType, uint index, uint x, uint y, int resX, int resY,
                              long seed, IntersectionLstItem *d_intersectionLst, RayIntersection *hairIntersectionLst) {

    thrust::default_random_engine rng(seed + index);
    thrust::uniform_real_distribution<float> uniDist;

    rng.discard(2 * index);

    float3 direction, color = make_float3(0.0f), yeFactor, xeFactor;
    for(int i = 0; i < SUPER_SAMPLING_2; i++) {
        yeFactor = ye * ((y + uniDist(rng)) / (float)resY - 0.5f);
        xeFactor = xe * ((x + uniDist(rng)) / (float)resX - 0.5f);

        direction = normalize(zeFactor + yeFactor + xeFactor);
        color += SUPER_SAMPLING_2_F * rayTracing(d_shapes, d_shapeSizes, lights, lightSize, backcolor, from, direction, 
                                                 rayInfo, d_colors, d_colorContributionType, index, d_intersectionLst,
                                                 hairIntersectionLst);
    }

    return color;
}

__device__
float3 stocasticHSSupersampling(int **d_shapes, uint *d_shapeSizes, Light *lights, uint lightSize, float3 backcolor, 
                                float3 xe, float3 ye, float3 zeFactor, float3 from, RayInfo* rayInfo, float3* d_colors, 
                                unsigned char *d_colorContributionType, uint index, uint x, uint y, int resX, int resY,
                                long seed, IntersectionLstItem *d_intersectionLst, RayIntersection *hairIntersectionLst) {

    uint hsIndex = index + seed;

    float3 direction, color = make_float3(0.0f), yeFactor, xeFactor;
    for(int i = 0; i < SUPER_SAMPLING_2; i++) {
        yeFactor = ye * ((y + haltonSequance(hsIndex + i, 3)) / (float)resY - 0.5f);
        xeFactor = xe * ((x + haltonSequance(hsIndex + i, 2)) / (float)resX - 0.5f);

        direction = normalize(zeFactor + yeFactor + xeFactor);

        color += SUPER_SAMPLING_2_F * rayTracing(d_shapes, d_shapeSizes, lights, lightSize, backcolor, from, direction, 
                                                 rayInfo, d_colors, d_colorContributionType, index, d_intersectionLst,
                                                 hairIntersectionLst);
    }

    return color;
}

__device__
float3 adaptiveStocasticSupersampling(int **d_shapes, uint *d_shapeSizes, Light *lights, uint lightSize, float3 backcolor, 
                                      float3 xe, float3 ye, float3 zeFactor, float3 from, RayInfo* rayInfo, float3* d_colors, 
                                      unsigned char *d_colorContributionType, uint index, uint x, uint y, int resX, int resY,
                                      long seed, int initNSamples, IntersectionLstItem *d_intersectionLst,
                                      RayIntersection *hairIntersectionLst) {

    thrust::default_random_engine rng(seed + index);
    thrust::uniform_real_distribution<float> uniDist;

    rng.discard(2 * index);

    float3 direction, color = make_float3(0.0f), yeFactor, xeFactor, tmp, ref;
    
    int samplingLimit = initNSamples;
    int step = 2;
    float factor = fmaxf(1.0f / samplingLimit, SUPER_SAMPLING_2_F);
    float difValue = 0.0f, threashold = initNSamples * 0.001f;

    ref = backcolor;

    for(int i = 0; i < SUPER_SAMPLING_2 && i < samplingLimit; i++) {
        yeFactor = ye * ((y + uniDist(rng)) / (float)resY - 0.5f);
        xeFactor = xe * ((x + uniDist(rng)) / (float)resX - 0.5f);

        direction = normalize(zeFactor + yeFactor + xeFactor);

        tmp = rayTracing(d_shapes, d_shapeSizes, lights, lightSize, backcolor, from, direction, 
                         rayInfo, d_colors, d_colorContributionType, index, d_intersectionLst, 
                         hairIntersectionLst);

        color += factor * tmp;

        difValue += length(ref - tmp);

        if(i == samplingLimit - 1 && samplingLimit * step <= SUPER_SAMPLING_2 && difValue >= threashold) {
            samplingLimit *= step;
            ref = color;
            color /= step;

            factor = 1.0f / samplingLimit;
            threashold *= step;
        }
    }

    //color debug output
    /*if(samplingLimit <= initNSamples) {
        return make_float3(0.76, 1, 0.96);
    } else if (samplingLimit <= 2 * initNSamples) {
        return make_float3(0, 0, 1);
    } else if(samplingLimit <= 4 * initNSamples) {
        return make_float3(0.6, 1, 0.6);
    } else if(samplingLimit <= 8 * initNSamples) {
        return make_float3(0, 1, 0);
    } else if(samplingLimit <= 16 * initNSamples) {
        return make_float3(1, 1, 0);
    } else if(samplingLimit <= 32 * initNSamples) {
        return make_float3(10, 0, 0);
    } else {
        return make_float3(0.5, 0, 0);
    }*/

    return color;
}


__global__
void drawScene(int **d_shapes, uint *d_shapeSizes, Light *lights, uint lightSize, float3 backcolor, int resX,
               int resY, float atDistance, float3 xe, float3 ye, float3 ze, float3 from, float3 *d_output,
               RayInfo *rayInfo, float3 *d_colors, unsigned char *d_colorContributionType, long seed,
               IntersectionLstItem *d_intersectionLst) {

    uint x = blockIdx.x * blockDim.x + threadIdx.x;
    uint y = blockIdx.y * blockDim.y + threadIdx.y;

    if(x >= resX || y >= resY) {
        return;
    }

    uint index = y * resX + x;
    
    #ifdef AT_HAIR
    RayIntersection hairIntersectionLst[HAIR_INTERSECTION_LST_SIZE];
    #else
    RayIntersection *hairIntersectionLst = nullptr;
    #endif

    d_output[index] = naiveSupersampling(d_shapes, d_shapeSizes, lights, lightSize, backcolor, xe, ye, ze, 
                                         from, rayInfo, d_colors, d_colorContributionType, index, x, y, resX, 
                                         resY, d_intersectionLst, hairIntersectionLst);
    
    /*d_output[index] = naiveRdmSupersampling(d_shapes, d_shapeSizes, lights, lightSize, backcolor, xe, ye, ze, 
                                            from, rayInfo, d_colors, d_colorContributionType, index, x, y, resX, 
                                            resY, seed, d_intersectionLst, hairIntersectionLst);*/

    /*d_output[index] = stocasticSupersampling(d_shapes, d_shapeSizes, lights, lightSize, backcolor, xe, ye, ze, 
                                             from, rayInfo, d_colors, d_colorContributionType, index, x, y, resX, 
                                             resY, seed, d_intersectionLst, hairIntersectionLst);*/

    /*d_output[index] = adaptiveStocasticSupersampling(d_shapes, d_shapeSizes, lights, lightSize, backcolor, xe, ye, ze, 
                                                     from, rayInfo, d_colors, d_colorContributionType, index, x, y, resX, 
                                                     resY, seed, 16, d_intersectionLst, hairIntersectionLst);*/

    /*d_output[index] = stocasticHSSupersampling(d_shapes, d_shapeSizes, lights, lightSize, backcolor, xe, ye, ze, 
                                               from, rayInfo, d_colors, d_colorContributionType, index, x, y, resX, 
                                               resY, seed, d_intersectionLst, hairIntersectionLst);*/

    #ifdef PRINT_N_INTERSECTIONS
    int rayHairIntersections = d_output[index].x;
    int nRays = d_output[index].y;
    d_output[index] = printRayHairIntersections(rayHairIntersections, backcolor, nRays);
    #endif

}


void deviceDrawScene(int **d_shapes, uint *d_shapeSizes, Light *lights, uint lightSize, float3 backcolor, 
                     int resX, int resY, float width, float height, float atDistance, float3 xe, float3 ye, 
                     float3 ze, float3 from, float3 *d_output, dim3 gridSize, dim3 blockSize, RayInfo *rayInfo,
                     float3 *d_colors, unsigned char *d_colorContributionType, long seed, 
                     IntersectionLstItem *d_intersectionLst) {
    
    
    ye *= height;
    xe *= width;
    ze = -ze * atDistance;
    drawScene<<<gridSize, blockSize>>>(d_shapes, d_shapeSizes, lights, lightSize, backcolor, resX, resY,
                                       atDistance, xe, ye, ze, from, d_output, rayInfo, d_colors, 
                                       d_colorContributionType, seed, d_intersectionLst);

}


float deviceBuildCylinderBVH(CylinderNode *bvh, uint nObjects, dim3 gridSize, dim3 blockSize, uint *mortonCodes, 
                             hipEvent_t &c_start, hipEvent_t &c_end, Cylinder *d_shapes, Matrix *d_matrixes, 
                             float3 *d_translations, uint *d_OBBIndexes, uint nOBBs) {
    float *areaVector, *costVector;
    int *lock, *nodeCounter;

    uint size = (2 * nObjects - 1) * sizeof(float); 
    checkCudaErrors(hipMalloc((void**) &areaVector, size));
    checkCudaErrors(hipMemset(areaVector, INT_MAX, size));

    checkCudaErrors(hipMalloc((void**) &costVector, size));
    checkCudaErrors(hipMemset(costVector, INT_MAX, size));

    size = (2 * nObjects - 1) * sizeof(int); 
    checkCudaErrors(hipMalloc((void**) &lock, size));
    checkCudaErrors(hipMemset(lock, 0, size));

    hipEventRecord(c_start);
    buildBVH<<<gridSize, blockSize>>>(bvh, nObjects, mortonCodes);

    computeBVHBB<<<gridSize, blockSize>>>(bvh, nObjects, lock, d_shapes, d_matrixes, 
                                          d_translations, d_OBBIndexes, nOBBs);

    nodeCounter = lock;
    size = nObjects * sizeof(int);
    checkCudaErrors(hipMemset(nodeCounter, 0, size));
    optimizeBVH<<<gridSize, blockSize>>>(bvh, nObjects, nodeCounter, areaVector, costVector);

    computeLeavesOBBs<<<gridSize, blockSize>>>(bvh, nObjects);
    hipEventRecord(c_end);

    hipEventSynchronize(c_end);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, c_start, c_end);

    checkCudaErrors(hipFree(lock));
    checkCudaErrors(hipFree(areaVector));
    checkCudaErrors(hipFree(costVector));
    checkCudaErrors(hipFree(mortonCodes));
    if(d_OBBIndexes != nullptr) {
        checkCudaErrors(hipFree(d_OBBIndexes));
    }

    return milliseconds / 1000.0f;
}

template <typename BVHNodeType, typename ShapeType>
float deviceBuildBVH(BVHNodeType *bvh, uint nObjects, dim3 gridSize, dim3 blockSize, uint *mortonCodes, 
                     hipEvent_t &c_start, hipEvent_t &c_end, ShapeType *d_shapes) {
    float *areaVector, *costVector;
    int *lock, *nodeCounter;

    uint size = (2 * nObjects - 1) * sizeof(float); 
    checkCudaErrors(hipMalloc((void**) &areaVector, size));
    checkCudaErrors(hipMemset(areaVector, INT_MAX, size));

    checkCudaErrors(hipMalloc((void**) &costVector, size));
    checkCudaErrors(hipMemset(costVector, INT_MAX, size));

    size = (2 * nObjects - 1) * sizeof(int); 
    checkCudaErrors(hipMalloc((void**) &lock, size));
    checkCudaErrors(hipMemset(lock, 0, size));

    hipEventRecord(c_start);
    buildBVH<<<gridSize, blockSize>>>(bvh, nObjects, mortonCodes);

    computeBVHBB<<<gridSize, blockSize>>>(bvh, nObjects, lock, d_shapes);

    nodeCounter = lock;
    size = nObjects * sizeof(int);
    checkCudaErrors(hipMemset(nodeCounter, 0, size));
    optimizeBVH<<<gridSize, blockSize>>>(bvh, nObjects, nodeCounter, areaVector, costVector);

    hipEventRecord(c_end);

    hipEventSynchronize(c_end);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, c_start, c_end);

    checkCudaErrors(hipFree(lock));
    checkCudaErrors(hipFree(areaVector));
    checkCudaErrors(hipFree(costVector));
    checkCudaErrors(hipFree(mortonCodes));

    return milliseconds / 1000.0f;
}

float deviceBuildSphereBVH(SphereNode *bvh, uint nObjects, dim3 gridSize, dim3 blockSize, uint *mortonCodes, 
                           hipEvent_t &c_start, hipEvent_t &c_end, Sphere *d_shapes) {

    return  deviceBuildBVH(bvh, nObjects, gridSize, blockSize, mortonCodes, c_start, c_end, d_shapes);
}

float deviceBuildTriangleBVH(TriangleNode *bvh, uint nObjects, dim3 gridSize, dim3 blockSize, uint *mortonCodes, 
                            hipEvent_t &c_start, hipEvent_t &c_end, Triangle *d_shapes) {

    return  deviceBuildBVH(bvh, nObjects, gridSize, blockSize, mortonCodes, c_start, c_end, d_shapes);
}

#endif