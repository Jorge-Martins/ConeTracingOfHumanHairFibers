#include "hip/hip_runtime.h"
#pragma once

#ifndef _RAYTRACER_KERNEL_CU_
#define _RAYTRACER_KERNEL_CU_

#include <hip/hip_vector_types.h>
#include <vector_functions.h>
#include <math_functions.h>

#include <cfloat>

#include "Scene.h"

__device__ float EPSILON = 1E-5f;

//size ray array
__device__ int const raysSize = 2 << (MAX_DEPTH - 1);

//reflection and refraction arrays 
__device__ int const sizeRRArrays = (2 << (MAX_DEPTH - 1)) - 1;

//size local array
__device__ int const raysPerPixel = (2 << MAX_DEPTH) - 1;

__device__
bool equal(float f1, float f2) {
	float diffAbs = abs(f1 - f2);
	return diffAbs < EPSILON;
}

__device__
bool intersection(Ray ray, RayIntersection *out, Plane plane) {
    float nDOTrdir = dot(plane.normal, ray.direction);

	if (equal(nDOTrdir, 0.0f)) {
		return false;
    }

	float nDOTr0 = dot(plane.normal, ray.origin);
	float t = -((nDOTr0 + plane.distance) / nDOTrdir);

	if (t < 0.0f) {
		return false;
    }

	if (out != nullptr) {
        out->distance = t;
		out->normal = plane.normal;
		out->point = ray.origin + t*ray.direction;
        out->shapeMaterial = plane.material;
		out->isEntering = nDOTrdir < 0.0f;
        
        out->point += out->normal * EPSILON;
	}
	return true;
}

__device__
bool intersection(Ray ray, RayIntersection *out, Triangle tri) {
    float normalDOTray = dot(tri.normal, ray.direction); 
    
    float3 h = cross(ray.direction, tri.e2);
	float a = dot(tri.e1, h);

    if (a > -EPSILON && a < EPSILON) {
		return false;
    }
	float f = 1.0f / a;
	float3 s = ray.origin - tri.vertices[0];
	float u = f * dot(s, h);

	if (u < 0.0 || u > 1.0) {
		return false;
    }
	float3 q = cross(s, tri.e1);
	float v = f * dot(ray.direction, q);

	if (v < 0.0 || u + v > 1.0) {
		return false;
    }
	
	float t = f * dot(tri.e2, q);

	if (t < 0) {
		return false;
    }

	if (out != nullptr) {
		out->distance = t;
		out->normal = tri.normal;
		out->point = ray.origin + t * ray.direction;
        out->shapeMaterial = tri.material;
		out->isEntering = normalDOTray < 0.0f;

        out->point += out->normal * EPSILON;
	}

	return true;
}

__device__
bool intersection(Ray ray, RayIntersection *out, Sphere sphere) {
    float d_2, r_2, b, root, t;

    float3 s_r = sphere.center - ray.origin;
    
    r_2 = sphere.r * sphere.r;
    d_2 = dot(s_r, s_r);

    if(equal(d_2, r_2)) {
        return false;
    }
    b = dot(ray.direction, s_r);

    if (d_2 > r_2 && b < 0.0f) {
        return false; 
    }

    root = b*b - d_2 + r_2;
    if(root < 0.0f) {
        return false;
    }

    t = min(b - sqrtf(root), b + sqrtf(root));

    if (out != nullptr) {
        out->point = ray.origin + ray.direction * t;
		out->normal = normalize((out->point - sphere.center) / sphere.r);

		bool entering = true;
		if (d_2 < r_2) {
			out->normal *= -1.0f;
			entering = false;
		}
        
        out->point += out->normal * EPSILON;
		out->shapeMaterial = sphere.material;
		out->distance = t;
        out->isEntering = entering;
	}

    return true;
}

__device__
bool infiniteCylinderIntersection(Ray ray, RayIntersection *out, Cylinder cylinder, float3 axis, float *inD, float *outD) {
    float3 r_c = ray.origin - cylinder.base;
    float r_2 = cylinder.radius * cylinder.radius;
    float3 n = cross(ray.direction, axis);

    float ln = length(n);

    // check if is parallel
    if(equal(ln, 0.0f)) {
        *inD = -1.0e21;
	    *outD = 1.0e21;
        return length(r_c - dot(r_c, axis) * axis) <= cylinder.radius;
    }
    n = normalize(n);

    float d = fabs(dot(r_c, n));

    if (d <= cylinder.radius) {
        float3 O = cross(r_c, axis);
    
        float t = -dot(O, n) / ln;
    
        O = normalize(cross(n, axis));

        float s = fabs(sqrtf(r_2 - d*d) / dot(ray.direction, O));

        *inD = t - s;
        *outD = t + s;

        return true;
    }

	return false;
}

__device__
bool intersection(Ray ray, RayIntersection *out, Cylinder cylinder) {
    float3 axis = normalize(cylinder.top - cylinder.base);
    float3 normal, point; 

    float baseDistance = -dot(-axis, cylinder.base);
    float topDistance = -dot(axis, cylinder.top);

    float dc, dw, t;
	float inD, outD;		/* Object  intersection dists.	*/
    //0 top, 1 side, 2 base
    short sideIn;
    short sideOut;

    if(!infiniteCylinderIntersection(ray, out, cylinder, axis, &inD, &outD)) {
        return false;
    }
    
    sideIn = sideOut = 1;

    /*	Intersect the ray with the bottom end-cap plane.		*/

	dc = dot(-axis, ray.direction);
    dw = dot(-axis, ray.origin) + baseDistance;

    if(dc == 0.0f) {		/* If parallel to bottom plane	*/
        if(dw >= 0.0f) {
            return false;
        }
    } else {
        t  = -dw / dc;
        if(dc >= 0.0f) {			    /* If far plane	*/
            if(t > inD && t < outD) {
                outD = t;
                sideOut = 2;
            }
            if(t < inD) {
                return false;
            }
        } else {				    /* If near plane	*/
            if(t > inD && t < outD) {
                inD	= t;
                sideIn = 2;
                
            }
            if(t > outD) {
                return false;
            }
        }
    }

/*	Intersect the ray with the top end-cap plane.			*/

    dc = dot(axis, ray.direction);
    dw = dot(axis, ray.origin) + topDistance;

	if(dc == 0.0f) {		/* If parallel to top plane	*/
	    if(dw >= 0.0f) {
            return false;
        }
	} else {
	    t  = - dw/dc;
	    if	(dc >= 0.0f) {			    /* If far plane	*/
		    if(t > inD && t < outD) {
                outD = t;
                sideOut = 0;
            }
		    if(t < inD) {
                return false;
            }
	    } else {				    /* If near plane	*/
		    if(t > inD && t < outD) {
                inD	= t;
                sideIn = 0;
                
            }
		    if(t > outD) {
                return false;
	        }
	    } 
    }

    if (inD < 0 && outD < 0) {
		return false;
    }

	if (inD < outD && inD > 0) {
		t = inD;
        point = ray.origin + t * ray.direction;

        if(sideIn == 0) {
            normal = axis;
        } else if(sideIn == 1) {
            float3 v1 = point - cylinder.base;
	        float3 v2 = dot(v1, axis) * axis;
	        normal = normalize(v1 - v2);
        } else {
            normal = -axis;
        }
        
    } else if (outD > 0) {
		t = outD;

        point = ray.origin + t * ray.direction;

        if(sideOut == 0) {
            normal = -axis;
        } else if(sideOut == 1) {
            float3 v1 = point - cylinder.base;
	        float3 v2 = dot(v1, axis) * axis;
	        normal = normalize(v2 - v1);
        } else {
            normal = axis;
        }
        
    } else {
        return false;
    }

    if (out != nullptr) {
        out->isEntering = dot(normal, ray.direction) < 0.0f;
        out->shapeMaterial = cylinder.material;
        out->distance = t;
        out->point = point;
        out->normal = normal;

        out->point += normal * EPSILON;
	}

    return true;
}

__device__
bool findShadow(int **d_shapes, size_t *d_shapeSizes, Ray feeler) {
    bool intersectionFound = false;
    for(size_t shapeType = 0; shapeType < nShapes; shapeType++) {
        for (size_t i = 0; i < d_shapeSizes[shapeType]; i++) {
            if(shapeType == sphereIndex) {
                Sphere *sphere = (Sphere*) d_shapes[shapeType];
                intersectionFound = intersection(feeler, nullptr, sphere[i]);

            } else if(shapeType == cylinderIndex) {
                Cylinder *cylinder = (Cylinder*) d_shapes[shapeType];
                intersectionFound = intersection(feeler, nullptr, cylinder[i]);

            } else if(shapeType == triangleIndex) {
                Triangle *triangle = (Triangle*) d_shapes[shapeType];
                intersectionFound = intersection(feeler, nullptr, triangle[i]);
            
            } else if(shapeType == planeIndex) {
                Plane *plane = (Plane*) d_shapes[shapeType];
                intersectionFound = intersection(feeler, nullptr, plane[i]);

            } else {
                return false;
            }

            if(intersectionFound) {
                return true;
            } 
	    }
    }

    return intersectionFound;
}

__device__
bool nearestIntersect(int **d_shapes, size_t *d_shapeSizes, Ray ray, RayIntersection *out) {
	RayIntersection minIntersect(FLT_MAX, make_float3(0.0f), make_float3(0.0f));
	bool minIntersectionFound = false, intersectionFound = false;

	RayIntersection curr = minIntersect;
    for(size_t shapeType = 0; shapeType < nShapes; shapeType++) {
        for (size_t i = 0; i < d_shapeSizes[shapeType]; i++) {
            if(shapeType == sphereIndex) {
                Sphere *sphere = (Sphere*) d_shapes[shapeType];
                intersectionFound = intersection(ray, &curr, sphere[i]);

            } else if(shapeType == cylinderIndex) {
                Cylinder *cylinder = (Cylinder*) d_shapes[shapeType];
                intersectionFound = intersection(ray, &curr, cylinder[i]);

            } else if(shapeType == triangleIndex) {
                Triangle *triangle = (Triangle*) d_shapes[shapeType];
                intersectionFound = intersection(ray, &curr, triangle[i]);
            
            } else if(shapeType == planeIndex) {
                Plane *plane = (Plane*) d_shapes[shapeType];
                intersectionFound = intersection(ray, &curr, plane[i]);

            } else {
                return false;
            }

		    if (intersectionFound) {
                if (curr.distance < minIntersect.distance) {
                    minIntersectionFound = true;
				    minIntersect = curr;
			    }
		    }
	    }
    }
    
	if (minIntersectionFound) {
		*out = minIntersect;
	}
	return minIntersectionFound;
}

__device__
void compensatePrecision(Ray &ray) {
	ray.origin += ray.direction * EPSILON;
}

__device__
int rayIndex(int index) {
    return index % raysSize;
}

__device__
float3 refract(float3 inDir, float3 normal, float eta) {
    float cosi = dot(-inDir, normal);
    float cost2 = 1.0f - eta * eta * (1.0f - cosi*cosi);
    float3 t = eta*inDir + ((eta*cosi - sqrt(abs(cost2))) * normal);

    if(cost2 > 0) {
        return t;
    } 

    return make_float3(0.0f);
}

__device__
float3 computeTransmissionDir(float3 inDir, float3 normal, float beforeIOR, float afterIOR) {
	return refract(inDir, normal, beforeIOR / afterIOR);
}

__device__
float3 rayTracing(int **d_shapes, size_t *d_shapeSizes, Light* lights, size_t lightSize, float3 backcolor, 
                 float3 rayOrigin, float3 rayDirection, Ray *ray, float3* locals,
                 float3* reflectionCols, float3* refractionCols, uint offset) {

    uint rayOffset = offset * raysSize;                 
    uint localsOffset = offset * raysPerPixel;
    uint rrOffset = offset * sizeRRArrays;
    Ray feeler = Ray();

    ray[rayOffset].update(rayOrigin, rayDirection);
    
    RayIntersection intersect;
    int level;
    float3 blackColor = make_float3(0.0f);
    
    for(int rayN = 0; rayN < raysPerPixel; rayN++) {
        //skip secundary rays that don't exist
        if(!ray[rayOffset + rayIndex(rayN)].exists) {
            if(rayN < sizeRRArrays) {
                reflectionCols[rrOffset + rayN] = blackColor;
                refractionCols[rrOffset + rayN] = blackColor;
                level = 2 * rayN;
                ray[rayOffset + rayIndex(level + 1)].exists = false;
                ray[rayOffset + rayIndex(level + 2)].exists = false;
            }
            continue;
        }

	    bool foundIntersect = nearestIntersect(d_shapes, d_shapeSizes, ray[rayOffset + rayIndex(rayN)], &intersect);

	    if (!foundIntersect) {
            if(rayN == 0) {
                return backcolor;
            }

            locals[localsOffset + rayN] = backcolor;

            if(rayN < sizeRRArrays) {
                reflectionCols[rrOffset + rayN] = blackColor;
                refractionCols[rrOffset + rayN] = blackColor;
                level = 2 * rayN;
                ray[rayOffset + rayIndex(level + 1)].exists = false;
                ray[rayOffset + rayIndex(level + 2)].exists = false;
            }
            continue;
        } 

        Material mat = intersect.shapeMaterial;
    
        // local illumination
	    locals[localsOffset + rayN] = blackColor;
	    for(size_t li = 0; li < lightSize; li++) {
		    float3 feelerDir = normalize(lights[li].position - intersect.point);
            feeler.update(intersect.point, feelerDir);
            
            bool inShadow = findShadow(d_shapes, d_shapeSizes, feeler);
                
		    if(!inShadow) {
                float3 reflectDir = reflect(-feelerDir, intersect.normal);
                float Lspec = powf(fmax(dot(reflectDir, -ray[rayOffset + rayIndex(rayN)].direction), 0.0f), 
                                         mat.shininess);
                float Ldiff = fmax(dot(feelerDir, intersect.normal), 0.0f);

			    
                locals[localsOffset + rayN] +=  (Ldiff * mat.color * mat.Kdiffuse + mat.color * Lspec * mat.Kspecular) * lights[li].color;
		    }
	    }
    
        if(rayN < sizeRRArrays) {
            reflectionCols[rrOffset + rayN] = blackColor;
            refractionCols[rrOffset + rayN] = blackColor;
            level = 2 * rayN;
            ray[rayOffset + rayIndex(level + 1)].exists = false;
            ray[rayOffset + rayIndex(level + 2)].exists = false;
            // reflection
            level = 2 * rayN + 1;
	        if(mat.Kspecular > 0.0f) {
                float3 reflectDir = reflect(ray[rayOffset + rayIndex(rayN)].direction, intersect.normal);
		        ray[rayOffset + rayIndex(level)].update(intersect.point, reflectDir);
                reflectionCols[rrOffset + rayN] = mat.color * mat.Kspecular;
            }

	        // transmission
            level = 2 * rayN + 2;
	        if(mat.transparency > 0.0f) {
		        float ior1, ior2;
		        if(intersect.isEntering) {
			        ior1 = 1.0f;
			        ior2 = mat.ior;
		        }
		        else {
			        ior1 = mat.ior;
			        ior2 = 1.0f;
		        }
		        float3 refractionDir = computeTransmissionDir(ray[rayOffset + rayIndex(rayN)].direction, intersect.normal, ior1, ior2);
		            
                if (!equal(length(refractionDir), 0.0f)) {
			        ray[rayOffset + rayIndex(level)].update(intersect.point, refractionDir);
                    refractionCols[rrOffset + rayN] = mat.color * mat.transparency;
                }
	        }
        }
    }

    int startLevel = sizeRRArrays - 1;
    int rrLevel = -2;

    if(MAX_DEPTH > 2) {
        rrLevel += 2 << (MAX_DEPTH - 2);
    }

    for(int i = startLevel; i >= 0 && i > rrLevel; i--) {
        level = 2 * i;       
        reflectionCols[rrOffset + i] *= locals[localsOffset + level + 1];
        refractionCols[rrOffset + i] *= locals[localsOffset + level + 2];
    }

    for(int i = rrLevel; i >= 0; i--) {
        level = 2 * i;
        locals[localsOffset + level + 1] += reflectionCols[rrOffset + level + 1] + refractionCols[rrOffset + level + 1];
        locals[localsOffset + level + 2] += reflectionCols[rrOffset + level + 2] + refractionCols[rrOffset + level + 2];
        
        reflectionCols[rrOffset + i] *= locals[localsOffset + level + 1];
        refractionCols[rrOffset + i] *= locals[localsOffset + level + 2];
    }

    return locals[localsOffset] + reflectionCols[rrOffset] + refractionCols[rrOffset];
}


__global__
void drawScene(int **d_shapes, size_t *d_shapeSizes, Light *lights, size_t lightSize, float3 backcolor, int resX,
               int resY, float width, float height, float atDistance, float3 xe, float3 ye, 
               float3 ze, float3 from, float3 *d_output, Ray* ray, float3* d_locals, 
               float3* d_reflectionCols, float3* d_refractionCols) {

    uint x = blockIdx.x * blockDim.x + threadIdx.x;
    uint y = blockIdx.y * blockDim.y + threadIdx.y;

    uint index = y * resX + x;

    float3 zeFactor = -ze * atDistance; 
	float3 yeFactor = ye * height * ((y + 0.5f) / (float)resY - 0.5f);
	float3 xeFactor = xe * width * ((x + 0.5f) / (float)resX - 0.5f);

	float3 direction = normalize(zeFactor + yeFactor + xeFactor);
	
    float3 color = rayTracing(d_shapes, d_shapeSizes, lights, lightSize, backcolor, from, direction, 
                              ray, d_locals, d_reflectionCols, d_refractionCols, index);
    
    
    d_output[index] = color;
   
}

__global__
void clearImage(float3 *d_output, float3 value, int resX) {
    uint x = blockIdx.x * blockDim.x + threadIdx.x;
    uint y = blockIdx.y * blockDim.y + threadIdx.y;

    uint index = y * resX + x;

    d_output[index] = value;
}

void deviceDrawScene(int **d_shapes, size_t *d_shapeSizes, Light* lights, size_t lightSize, float3 backcolor, 
                     int resX, int resY, float width, float height, float atDistance, float3 xe, float3 ye, 
                     float3 ze, float3 from, float3 *d_output, dim3 gridSize, dim3 blockSize, Ray* ray,
                     float3* d_locals, float3* d_reflectionCols, float3* d_refractionCols) {

    /*if(SUPER_SAMPLING > 1) {
        dim3 clearGridSize = dim3(gridSize.x / SUPER_SAMPLING, gridSize.y / SUPER_SAMPLING, gridSize.z);
        clearImage<<<clearGridSize, blockSize>>>(d_output, make_float3(0.0f), resX);

        hipDeviceSynchronize();
    }*/

    drawScene<<<gridSize, blockSize>>>(d_shapes, d_shapeSizes, lights, lightSize, backcolor, resX, resY,
                                       width, height, atDistance, xe, ye, ze, from, d_output, ray,
                                       d_locals, d_reflectionCols, d_refractionCols);

}


#endif